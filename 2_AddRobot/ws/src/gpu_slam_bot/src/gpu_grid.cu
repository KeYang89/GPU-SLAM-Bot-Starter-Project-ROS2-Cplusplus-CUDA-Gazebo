#include "hip/hip_runtime.h"
#include "gpu_slam_bot/gpu_grid.hpp"
#include <hip/hip_runtime.h>
#include <cmath>
#include <stdexcept>

using namespace gpu_slam_bot;

static inline void checkCuda(hipError_t err, const char* msg){
  if (err != hipSuccess) throw std::runtime_error(std::string(msg)+": "+hipGetErrorString(err));
}

__device__ __forceinline__
int idx(int x, int y, int W){ return y*W + x; }

__device__ float clampf(float v, float lo, float hi){ return fminf(hi, fmaxf(lo, v)); }

// DDA ray march updating free cells; final hit cell marked occupied.
__global__ void integrate_kernel(float* grid, int W, int H,
                                 float res, float ox, float oy,
                                 float l_free, float l_occ, float lmin, float lmax,
                                 const float* ranges, const float* angles,
                                 int nbeams,
                                 float rx, float ry, float ryaw, float rmax)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= nbeams) return;

  float r = ranges[i];
  float a = angles[i] + ryaw;
  bool hit = (r > 0.0f && r < rmax*0.999f);
  float use_r = hit ? r : rmax;

  // Endpoint in world
  float ex = rx + use_r * cosf(a);
  float ey = ry + use_r * sinf(a);

  // Ray start in world
  float sx = rx;
  float sy = ry;

  // Convert to grid coords
  int gx0 = (int)floorf((sx - ox) / res);
  int gy0 = (int)floorf((sy - oy) / res);
  int gx1 = (int)floorf((ex - ox) / res);
  int gy1 = (int)floorf((ey - oy) / res);

  // DDA setup
  int dx = abs(gx1 - gx0), sxg = gx0 < gx1 ? 1 : -1;
  int dy = abs(gy1 - gy0), syg = gy0 < gy1 ? 1 : -1;
  int err = (dx > dy ? dx : -dy)/2; // Bresenham-like integer DDA

  int x = gx0, y = gy0;
  while (true) {
    if (x>=0 && x<W && y>=0 && y<H) {
      int id = idx(x,y,W);
      float v = atomicAdd(&grid[id], l_free); // freespace along ray
      // clamp after add (benign race; acceptable for mapping)
      grid[id] = clampf(v + 0.0f, lmin, lmax);
    }
    if (x==gx1 && y==gy1) break;
    int e2 = err;
    if (e2 > -dx) { err -= dy; x += sxg; }
    if (e2 <  dy) { err += dx; y += syg; }
  }

  // Mark endpoint occupied if it was an actual hit
  if (hit && gx1>=0 && gx1<W && gy1>=0 && gy1<H) {
    int id = idx(gx1, gy1, W);
    float v = atomicAdd(&grid[id], l_occ);
    grid[id] = clampf(v + 0.0f, lmin, lmax);
  }
}

namespace gpu_slam_bot {

GpuGrid::GpuGrid(const GridParams &p): params_(p) {
  size_t bytes = (size_t)p.width * p.height * sizeof(float);
  checkCuda(hipMalloc(&d_grid_, bytes), "hipMalloc grid");
  checkCuda(hipMemset(d_grid_, 0, bytes), "hipMemset grid");
}

GpuGrid::~GpuGrid(){ if (d_grid_) hipFree(d_grid_); }

void GpuGrid::integrateScan(const std::vector<float> &ranges,
                            const std::vector<float> &angles,
                            float rx, float ry, float ryaw,
                            float rmax)
{
  int n = (int)ranges.size();
  if ((int)angles.size() != n) throw std::runtime_error("ranges/angles size mismatch");

  float *d_ranges=nullptr, *d_angles=nullptr;
  checkCuda(hipMalloc(&d_ranges, n*sizeof(float)), "malloc ranges");
  checkCuda(hipMalloc(&d_angles, n*sizeof(float)), "malloc angles");
  checkCuda(hipMemcpy(d_ranges, ranges.data(), n*sizeof(float), hipMemcpyHostToDevice), "cpy ranges");
  checkCuda(hipMemcpy(d_angles, angles.data(), n*sizeof(float), hipMemcpyHostToDevice), "cpy angles");

  int threads = 256;
  int blocks = (n + threads - 1) / threads;

  integrate_kernel<<<blocks, threads>>>(d_grid_, params_.width, params_.height,
    params_.resolution, params_.origin_x, params_.origin_y,
    params_.l_free, params_.l_occ, params_.l_min, params_.l_max,
    d_ranges, d_angles, n, rx, ry, ryaw, rmax);

  checkCuda(hipGetLastError(), "kernel launch");
  hipFree(d_ranges); hipFree(d_angles);
}

static inline int8_t logOddsToOcc(float l){
  // p = 1 - 1/(1+exp(l)) = exp(l)/(1+exp(l))
  float p = 1.0f - 1.0f/(1.0f + std::exp(l));
  int v = (int)std::round(p * 100.0f);
  if (v < 0) v = 0; if (v > 100) v = 100; return (int8_t)v;
}

void GpuGrid::downloadToOcc(std::vector<int8_t> &occ_out) const {
  size_t N = (size_t)params_.width * params_.height;
  std::vector<float> h(N);
  checkCuda(hipMemcpy(h.data(), d_grid_, N*sizeof(float), hipMemcpyDeviceToHost), "download grid");
  occ_out.resize(N);
  for (size_t i=0;i<N;++i) occ_out[i] = logOddsToOcc(h[i]);
}

} // namespace gpu_slam_bot