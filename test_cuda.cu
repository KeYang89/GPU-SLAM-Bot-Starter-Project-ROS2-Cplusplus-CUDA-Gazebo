#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    std::cout << "CUDA devices: " << deviceCount << std::endl;
    return 0;
}
